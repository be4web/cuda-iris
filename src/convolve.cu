#include "hip/hip_runtime.h"
extern "C" {
#include "convolve.h"
#include <stdint.h>
}

// http://developer.download.nvidia.com/assets/cuda/files/convolutionSeparable.pdf

/*!
 * USR: user specified identifier for this convolution
 * IN_T: input pixel type
 * OUT_T: output pixel type
 * MTX_T: matrix scalar type (this type is used for intermediate results!)
 * MTX_S: matrix size
 * BLOCK_W: block width (block x dimension); note: BLOCK_W must be >= (MTX_S / 2) to fill apron
 * BLOCK_H: block height (block y dimension)
 * STEPS: number of convolution steps performed (number of output pixels written) by each thread
 */
#define DECL_CU_CONVOLUTION_ROW(USR, IN_T, OUT_T, MTX_T, MTX_S, BLOCK_W, BLOCK_H, STEPS)                               \
__global__ void convol_row_k_##USR##MTX_S(IN_T *in, OUT_T *out, MTX_T *mtx, MTX_T div, int img_w)                      \
{                                                                                                                      \
    __shared__ IN_T bl_d[(BLOCK_H)][((STEPS) + 2) * (BLOCK_W)];                                                        \
                                                                                                                       \
    /* offset to left edge of apron */                                                                                 \
    const int off_x = (blockIdx.x * (STEPS) - 1) * (BLOCK_W) + threadIdx.x;                                            \
    const int off_y = blockIdx.y * (BLOCK_H) + threadIdx.y;                                                            \
                                                                                                                       \
    in += off_y * img_w + off_x;                                                                                       \
    out += off_y * img_w + off_x;                                                                                      \
                                                                                                                       \
    /* left apron */                                                                                                   \
    bl_d[threadIdx.y][threadIdx.x] = (off_x >= 0) ? in[0] : in[-off_x];                                                \
                                                                                                                       \
    /* main data */                                                                                                    \
_Pragma("unroll")                                                                                                      \
    for (int i = 1; i <= (STEPS); i++)                                                                                 \
        bl_d[threadIdx.y][threadIdx.x + i * (BLOCK_W)] = in[i * (BLOCK_W)];                                            \
                                                                                                                       \
    /* right apron */                                                                                                  \
    bl_d[threadIdx.y][threadIdx.x + ((STEPS) + 1) * (BLOCK_W)] = (img_w - off_x > ((STEPS) + 1) * (BLOCK_W)) ?         \
                                                             in[((STEPS) + 1) * (BLOCK_W)] : in[img_w - off_x - 1];    \
                                                                                                                       \
    __syncthreads();                                                                                                   \
                                                                                                                       \
_Pragma("unroll")                                                                                                      \
    for (int i = 1; i <= (STEPS); i++) {                                                                               \
        MTX_T sum = 0;                                                                                                 \
                                                                                                                       \
_Pragma("unroll")                                                                                                      \
        for (int j = -(MTX_S / 2); j <= ((MTX_S - 1) / 2); j++)                                                        \
            sum += mtx[(MTX_S / 2) + j] *  bl_d[threadIdx.y][threadIdx.x + i * (BLOCK_W) + j];                         \
                                                                                                                       \
        out[i * (BLOCK_W)] = sum / div;                                                                                \
    }                                                                                                                  \
}                                                                                                                      \
                                                                                                                       \
static void cu_convolve_row_##USR##MTX_S(void *gm_in, void *gm_out, void *gm_mtx, MTX_T div, int img_w, int img_h)     \
{                                                                                                                      \
    dim3 blocks(img_w / ((BLOCK_W) * (STEPS)), img_h / (BLOCK_H));                                                     \
    dim3 threads((BLOCK_W), (BLOCK_H));                                                                                \
                                                                                                                       \
    convol_row_k_##USR##MTX_S<<<blocks, threads>>>((IN_T *)gm_in, (OUT_T *)gm_out, (MTX_T *)gm_mtx, div, img_w);       \
}

/*!
 * USR: user specified identifier for this convolution
 * IN_T: input pixel type
 * OUT_T: output pixel type
 * MTX_T: matrix scalar type (this type is used for intermediate results!)
 * MTX_S: matrix size
 * BLOCK_W: block width (block x dimension)
 * BLOCK_H: block height (block y dimension); note: BLOCK_H must be >= (MTX_S / 2) to fill apron
 * STEPS: number of convolution steps performed (number of output pixels written) by each thread
 */
#define DECL_CU_CONVOLUTION_COL(USR, IN_T, OUT_T, MTX_T, MTX_S, BLOCK_W, BLOCK_H, STEPS)                               \
__global__ void convol_col_k_##USR##MTX_S(IN_T *in, OUT_T *out, MTX_T *mtx, MTX_T div, int img_w, int img_h)           \
{                                                                                                                      \
    __shared__ IN_T bl_d[(BLOCK_W)][((STEPS) + 2) * (BLOCK_H)]; /* +1 */                                               \
                                                                                                                       \
    /* offset to left edge of apron */                                                                                 \
    const int off_x = blockIdx.x * (BLOCK_W) + threadIdx.x;                                                            \
    const int off_y = (blockIdx.y * (STEPS) - 1) * (BLOCK_H) + threadIdx.y;                                            \
                                                                                                                       \
    in += off_y * img_w + off_x;                                                                                       \
    out += off_y * img_w + off_x;                                                                                      \
                                                                                                                       \
    /* upper apron */                                                                                                  \
    bl_d[threadIdx.x][threadIdx.y] = (off_y >= 0) ? in[0] : in[-off_y * img_w];                                        \
                                                                                                                       \
    /* main data */                                                                                                    \
_Pragma("unroll")                                                                                                      \
    for (int i = 1; i <= (STEPS); i++)                                                                                 \
        bl_d[threadIdx.x][threadIdx.y + i * (BLOCK_H)] = in[i * (BLOCK_H) * img_w];                                    \
                                                                                                                       \
    /* lower apron */                                                                                                  \
    bl_d[threadIdx.x][threadIdx.y + ((STEPS) + 1) * (BLOCK_H)] = (img_h - off_y > ((STEPS) + 1) * (BLOCK_H)) ?         \
                                              in[((STEPS) + 1) * (BLOCK_H) * img_w] : in[(img_h - off_y - 1) * img_w]; \
                                                                                                                       \
    __syncthreads();                                                                                                   \
                                                                                                                       \
_Pragma("unroll")                                                                                                      \
    for (int i = 1; i <= (STEPS); i++) {                                                                               \
        MTX_T sum = 0;                                                                                                 \
                                                                                                                       \
_Pragma("unroll")                                                                                                      \
        for (int j = -(MTX_S / 2); j <= ((MTX_S - 1) / 2); j++)                                                        \
            sum += mtx[(MTX_S / 2) + j] *  bl_d[threadIdx.x][threadIdx.y + i * (BLOCK_H) + j];                         \
                                                                                                                       \
        out[i * (BLOCK_H) * img_w] = sum / div;                                                                        \
    }                                                                                                                  \
}                                                                                                                      \
                                                                                                                       \
static void cu_convolve_col_##USR##MTX_S(void *gm_in, void *gm_out, void *gm_mtx, MTX_T div, int img_w, int img_h)     \
{                                                                                                                      \
    dim3 blocks(img_w / (BLOCK_W), img_h / ((BLOCK_H) * (STEPS)));                                                     \
    dim3 threads((BLOCK_W), (BLOCK_H));                                                                                \
                                                                                                                       \
    convol_col_k_##USR##MTX_S<<<blocks, threads>>>((IN_T *)gm_in, (OUT_T *)gm_out, (MTX_T *)gm_mtx, div, img_w, img_h);\
}

#define WARP_SIZE 32

DECL_CU_CONVOLUTION_ROW(gauss, uint8_t, uint8_t, int, 3, 1, (WARP_SIZE / 1), 8)
DECL_CU_CONVOLUTION_COL(gauss, uint8_t, uint8_t, int, 3, (WARP_SIZE / 1), 1, 8)

DECL_CU_CONVOLUTION_ROW(gauss, uint8_t, uint8_t, int, 5, 2, (WARP_SIZE / 2), 8)
DECL_CU_CONVOLUTION_COL(gauss, uint8_t, uint8_t, int, 5, (WARP_SIZE / 2), 2, 8)

DECL_CU_CONVOLUTION_ROW(gauss, uint8_t, uint8_t, int, 7, 4, (WARP_SIZE / 4), 4)
DECL_CU_CONVOLUTION_COL(gauss, uint8_t, uint8_t, int, 7, (WARP_SIZE / 4), 4, 4)

DECL_CU_CONVOLUTION_ROW(gauss, uint8_t, uint8_t, int, 9, 4, (WARP_SIZE / 4), 4)
DECL_CU_CONVOLUTION_COL(gauss, uint8_t, uint8_t, int, 9, (WARP_SIZE / 4), 4, 4)

DECL_CU_CONVOLUTION_ROW(gauss, uint8_t, uint8_t, int, 11, 6, (WARP_SIZE / 6), 4)
DECL_CU_CONVOLUTION_COL(gauss, uint8_t, uint8_t, int, 11, (WARP_SIZE / 6), 6, 4)

__constant__ int gauss_mtx_3[3] = { 1, 2, 1 };
__constant__ int gauss_mtx_5[5] = { 1, 4, 6, 4, 1 };
__constant__ int gauss_mtx_7[7] = { 1, 6, 15, 20, 15, 6, 1 };
__constant__ int gauss_mtx_9[9] = { 1, 8, 28, 56, 70, 56, 28, 8, 1 };
__constant__ int gauss_mtx_11[11] = { 1, 10, 45, 120, 210, 252, 210, 120, 45, 10, 1 };

extern "C" int cu_gauss_filter(int rad, int img_w, int img_h, void *gm_in, void *gm_out, void *gm_tmp)
{
    void *mtx;

    switch (rad) {
        case 3:
            hipGetSymbolAddress(&mtx, HIP_SYMBOL(gauss_mtx_3));
            cu_convolve_row_gauss3(gm_in, gm_tmp, mtx, 4, img_w, img_h);
            cu_convolve_col_gauss3(gm_tmp, gm_out, mtx, 4, img_w, img_h);
            break;

        case 5:
            hipGetSymbolAddress(&mtx, HIP_SYMBOL(gauss_mtx_5));
            cu_convolve_row_gauss5(gm_in, gm_tmp, mtx, 16, img_w, img_h);
            cu_convolve_col_gauss5(gm_tmp, gm_out, mtx, 16, img_w, img_h);
            break;

        case 7:
            hipGetSymbolAddress(&mtx, HIP_SYMBOL(gauss_mtx_7));
            cu_convolve_row_gauss7(gm_in, gm_tmp, mtx, 64, img_w, img_h);
            cu_convolve_col_gauss7(gm_tmp, gm_out, mtx, 64, img_w, img_h);
            break;

        case 9:
            hipGetSymbolAddress(&mtx, HIP_SYMBOL(gauss_mtx_9));
            cu_convolve_row_gauss9(gm_in, gm_tmp, mtx, 256, img_w, img_h);
            cu_convolve_col_gauss9(gm_tmp, gm_out, mtx, 256, img_w, img_h);
            break;

        case 11:
            hipGetSymbolAddress(&mtx, HIP_SYMBOL(gauss_mtx_11));
            cu_convolve_row_gauss11(gm_in, gm_tmp, mtx, 1024, img_w, img_h);
            cu_convolve_col_gauss11(gm_tmp, gm_out, mtx, 1024, img_w, img_h);
            break;

        default:
            return -1;
    }

    return 0;
}

DECL_CU_CONVOLUTION_ROW(gaussf, float, float, float, 11, 6, (WARP_SIZE / 6), 4)
DECL_CU_CONVOLUTION_COL(gaussf, float, float, float, 11, (WARP_SIZE / 6), 6, 4)

__constant__ float gauss_mtx_f11[11] = { 1.0, 10.0, 45.0, 120.0, 210.0, 252.0, 210.0, 120.0, 45.0, 10.0, 1.0 };

extern "C" void cu_gauss_filter_f11(int img_w, int img_h, void *gm_in, void *gm_out, void *gm_tmp)
{
    void *mtx;

    hipGetSymbolAddress(&mtx, HIP_SYMBOL(gauss_mtx_f11));
    cu_convolve_row_gaussf11(gm_in, gm_tmp, mtx, 1024.0, img_w, img_h);
    cu_convolve_col_gaussf11(gm_tmp, gm_out, mtx, 1024.0, img_w, img_h);
}

DECL_CU_CONVOLUTION_ROW(sobel, uint8_t, int16_t, int, 3, 1, (WARP_SIZE / 1), 8);
DECL_CU_CONVOLUTION_COL(sobel, int16_t, int16_t, int, 3, (WARP_SIZE / 1), 1, 8);

__constant__ int sobel_mtx_1[3] = { -1, 0, 1 };
__constant__ int sobel_mtx_2[3] = { 1, 2, 1 };

extern "C" int cu_sobel_filter(int img_w, int img_h, void *gm_in, void *gm_hori, void *gm_vert, void *gm_tmp)
{
    void *mtx1, *mtx2;

    hipGetSymbolAddress(&mtx1, HIP_SYMBOL(sobel_mtx_1));
    hipGetSymbolAddress(&mtx2, HIP_SYMBOL(sobel_mtx_2));

    cu_convolve_row_sobel3(gm_in, gm_tmp, mtx1, 1, img_w, img_h);
    cu_convolve_col_sobel3(gm_tmp, gm_hori, mtx2, 1, img_w, img_h);

    cu_convolve_row_sobel3(gm_in, gm_tmp, mtx2, 1, img_w, img_h);
    cu_convolve_col_sobel3(gm_tmp, gm_vert, mtx1, 1, img_w, img_h);

    return 0;
}
