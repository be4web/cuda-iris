#include "hip/hip_runtime.h"
extern "C" {
#include "convolve.h"
#include <stdint.h>
}

/****** PARAMETERS ********/
#define MARGIN 20
// needs to be a power of 2 as well
#define STEP_SIZE 2

// diff should be a multiple of 2 to exploit warp size efficiently (> 32)
#define MIN_RAD 6
#define MAX_RAD 262
#define PI 3.141592654

#define THREAD_COUNT (MAX_RAD - MIN_RAD)
#define WARP_SIZE 32

texture<float, hipTextureType2D> phi_tex;
texture<float, hipTextureType2D> abs_tex;

/*
 * Warp reduce sum: http://devblogs.nvidia.com/parallelforall/faster-parallel-reductions-kepler/
 **/
__inline__ __device__
int warpReduceSum(int val) {
  for (int offset = warpSize / 2; offset > 0; offset /= 2)
    val += __shfl_down(val,offset);
  return val;
}

__inline__ __device__
int blockReduceSum(int val) {

  static __shared__ int shared[32]; // Shared mem for 32 partial sums
  int lane = threadIdx.x % warpSize;
  int wid = threadIdx.x / warpSize;

  val = warpReduceSum(val);     // Each warp performs partial reduction

  if (lane==0) shared[wid]=val; // Write reduced value to shared memory

  __syncthreads();              // Wait for all partial reductions

  //read from shared memory only if that warp existed
  val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : 0;

  if (wid==0) val = warpReduceSum(val); //Final reduce within first warp

  return val;
}

__global__ void hough_transform(int *result) {

    //__shared__ int radius_acc[32];

    float dx, dy;
    int radius_index = threadIdx.x;
    float rad = MIN_RAD + radius_index + STEP_SIZE;
    int x = MARGIN + blockIdx.x;
    int y = MARGIN + blockIdx.y;
    int width = MARGIN + gridDim.x;


    int local_acc = int(0);

    /*
    int phi;
    for (phi = 0; phi < 240; phi++) {
        float phi_f = (float)phi * PI / 120.0;

        dx = rad * cos(phi_f);
        dy = rad * sin(phi_f);

        if (tex2D(abs_tex, x + dx, y + dy) > 20) {
            float grad = PI + tex2D(phi_tex, x + dx, y + dy) - phi_f;
            if (-PI / 12.0 < grad && grad < PI / 12.0)
                local_acc++;
        }
    }
    */

    dx = rad * 0.9996573250;
    dy = rad * 0.0261769483;


    {

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 0.02617993877991494;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }

    dx = rad * 0.9986295348;
    dy = rad * 0.0523359562;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 0.05235987755982988;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.9969173337;
    dy = rad * 0.0784590957;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 0.07853981633974483;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.9945218954;
    dy = rad * 0.1045284633;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 0.10471975511965977;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.9914448614;
    dy = rad * 0.1305261922;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 0.1308996938995747;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.9876883406;
    dy = rad * 0.1564344650;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 0.15707963267948966;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.9832549076;
    dy = rad * 0.1822355255;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 0.1832595714594046;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.9781476007;
    dy = rad * 0.2079116908;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 0.20943951023931956;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.9723699204;
    dy = rad * 0.2334453639;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 0.2356194490192345;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.9659258263;
    dy = rad * 0.2588190451;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 0.26179938779914946;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.9588197349;
    dy = rad * 0.2840153447;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 0.2879793265790644;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.9510565163;
    dy = rad * 0.3090169944;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 0.3141592653589793;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.9426414911;
    dy = rad * 0.3338068592;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 0.34033920413889424;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.9335804265;
    dy = rad * 0.3583679495;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 0.36651914291880916;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.9238795325;
    dy = rad * 0.3826834324;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 0.3926990816987241;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.9135454576;
    dy = rad * 0.4067366431;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 0.418879020478639;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.9025852843;
    dy = rad * 0.4305110968;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 0.44505895925855393;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.8910065242;
    dy = rad * 0.4539904997;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 0.47123889803846886;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.8788171127;
    dy = rad * 0.4771587603;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 0.4974188368183838;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.8660254038;
    dy = rad * 0.5000000000;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 0.5235987755982987;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.8526401644;
    dy = rad * 0.5224985647;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 0.5497787143782137;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.8386705679;
    dy = rad * 0.5446390350;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 0.5759586531581287;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.8241261886;
    dy = rad * 0.5664062369;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 0.6021385919380436;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.8090169944;
    dy = rad * 0.5877852523;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 0.6283185307179586;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.7933533403;
    dy = rad * 0.6087614290;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 0.6544984694978736;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.7771459615;
    dy = rad * 0.6293203910;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 0.6806784082777886;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.7604059656;
    dy = rad * 0.6494480483;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 0.7068583470577036;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.7431448255;
    dy = rad * 0.6691306064;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 0.7330382858376185;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.7253743710;
    dy = rad * 0.6883545757;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 0.7592182246175335;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.7071067812;
    dy = rad * 0.7071067812;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 0.7853981633974485;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.6883545757;
    dy = rad * 0.7253743710;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 0.8115781021773635;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.6691306064;
    dy = rad * 0.7431448255;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 0.8377580409572785;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.6494480483;
    dy = rad * 0.7604059656;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 0.8639379797371934;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.6293203910;
    dy = rad * 0.7771459615;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 0.8901179185171084;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.6087614290;
    dy = rad * 0.7933533403;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 0.9162978572970234;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.5877852523;
    dy = rad * 0.8090169944;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 0.9424777960769384;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.5664062369;
    dy = rad * 0.8241261886;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 0.9686577348568534;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.5446390350;
    dy = rad * 0.8386705679;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 0.9948376736367683;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.5224985647;
    dy = rad * 0.8526401644;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 1.0210176124166832;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.5000000000;
    dy = rad * 0.8660254038;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 1.047197551196598;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.4771587603;
    dy = rad * 0.8788171127;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 1.073377489976513;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.4539904997;
    dy = rad * 0.8910065242;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 1.0995574287564278;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.4305110968;
    dy = rad * 0.9025852843;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 1.1257373675363427;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.4067366431;
    dy = rad * 0.9135454576;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 1.1519173063162575;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.3826834324;
    dy = rad * 0.9238795325;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 1.1780972450961724;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.3583679495;
    dy = rad * 0.9335804265;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 1.2042771838760873;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.3338068592;
    dy = rad * 0.9426414911;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 1.2304571226560022;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.3090169944;
    dy = rad * 0.9510565163;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 1.256637061435917;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.2840153447;
    dy = rad * 0.9588197349;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 1.282817000215832;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.2588190451;
    dy = rad * 0.9659258263;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 1.3089969389957468;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.2334453639;
    dy = rad * 0.9723699204;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 1.3351768777756616;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.2079116908;
    dy = rad * 0.9781476007;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 1.3613568165555765;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.1822355255;
    dy = rad * 0.9832549076;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 1.3875367553354914;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.1564344650;
    dy = rad * 0.9876883406;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 1.4137166941154062;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.1305261922;
    dy = rad * 0.9914448614;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 1.439896632895321;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.1045284633;
    dy = rad * 0.9945218954;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 1.466076571675236;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.0784590957;
    dy = rad * 0.9969173337;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 1.4922565104551508;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.0523359562;
    dy = rad * 0.9986295348;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 1.5184364492350657;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.0261769483;
    dy = rad * 0.9996573250;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 1.5446163880149806;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.0000000000;
    dy = rad * 1.0000000000;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 1.5707963267948954;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.0261769483;
    dy = rad * 0.9996573250;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 1.5969762655748103;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.0523359562;
    dy = rad * 0.9986295348;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 1.6231562043547252;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.0784590957;
    dy = rad * 0.9969173337;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 1.64933614313464;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.1045284633;
    dy = rad * 0.9945218954;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 1.675516081914555;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.1305261922;
    dy = rad * 0.9914448614;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 1.7016960206944698;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.1564344650;
    dy = rad * 0.9876883406;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 1.7278759594743847;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.1822355255;
    dy = rad * 0.9832549076;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 1.7540558982542995;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.2079116908;
    dy = rad * 0.9781476007;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 1.7802358370342144;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.2334453639;
    dy = rad * 0.9723699204;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 1.8064157758141293;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.2588190451;
    dy = rad * 0.9659258263;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 1.8325957145940441;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.2840153447;
    dy = rad * 0.9588197349;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 1.858775653373959;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.3090169944;
    dy = rad * 0.9510565163;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 1.8849555921538739;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.3338068592;
    dy = rad * 0.9426414911;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 1.9111355309337887;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.3583679495;
    dy = rad * 0.9335804265;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 1.9373154697137036;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.3826834324;
    dy = rad * 0.9238795325;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 1.9634954084936185;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.4067366431;
    dy = rad * 0.9135454576;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 1.9896753472735333;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.4305110968;
    dy = rad * 0.9025852843;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 2.015855286053448;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.4539904997;
    dy = rad * 0.8910065242;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 2.0420352248333633;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.4771587603;
    dy = rad * 0.8788171127;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 2.0682151636132784;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.5000000000;
    dy = rad * 0.8660254038;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 2.0943951023931935;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.5224985647;
    dy = rad * 0.8526401644;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 2.1205750411731086;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.5446390350;
    dy = rad * 0.8386705679;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 2.1467549799530237;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.5664062369;
    dy = rad * 0.8241261886;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 2.1729349187329388;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.5877852523;
    dy = rad * 0.8090169944;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 2.199114857512854;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.6087614290;
    dy = rad * 0.7933533403;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 2.225294796292769;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.6293203910;
    dy = rad * 0.7771459615;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 2.251474735072684;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.6494480483;
    dy = rad * 0.7604059656;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 2.277654673852599;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.6691306064;
    dy = rad * 0.7431448255;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 2.303834612632514;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.6883545757;
    dy = rad * 0.7253743710;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 2.3300145514124293;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.7071067812;
    dy = rad * 0.7071067812;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 2.3561944901923444;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.7253743710;
    dy = rad * 0.6883545757;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 2.3823744289722595;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.7431448255;
    dy = rad * 0.6691306064;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 2.4085543677521746;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.7604059656;
    dy = rad * 0.6494480483;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 2.4347343065320897;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.7771459615;
    dy = rad * 0.6293203910;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 2.4609142453120048;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.7933533403;
    dy = rad * 0.6087614290;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 2.48709418409192;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.8090169944;
    dy = rad * 0.5877852523;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 2.513274122871835;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.8241261886;
    dy = rad * 0.5664062369;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 2.53945406165175;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.8386705679;
    dy = rad * 0.5446390350;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 2.565634000431665;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.8526401644;
    dy = rad * 0.5224985647;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 2.59181393921158;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.8660254038;
    dy = rad * 0.5000000000;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 2.6179938779914953;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.8788171127;
    dy = rad * 0.4771587603;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 2.6441738167714104;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.8910065242;
    dy = rad * 0.4539904997;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 2.6703537555513255;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.9025852843;
    dy = rad * 0.4305110968;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 2.6965336943312406;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.9135454576;
    dy = rad * 0.4067366431;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 2.7227136331111557;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.9238795325;
    dy = rad * 0.3826834324;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 2.7488935718910708;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.9335804265;
    dy = rad * 0.3583679495;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 2.775073510670986;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.9426414911;
    dy = rad * 0.3338068592;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 2.801253449450901;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.9510565163;
    dy = rad * 0.3090169944;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 2.827433388230816;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.9588197349;
    dy = rad * 0.2840153447;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 2.853613327010731;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.9659258263;
    dy = rad * 0.2588190451;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 2.879793265790646;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.9723699204;
    dy = rad * 0.2334453639;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 2.9059732045705613;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.9781476007;
    dy = rad * 0.2079116908;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 2.9321531433504764;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.9832549076;
    dy = rad * 0.1822355255;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 2.9583330821303915;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.9876883406;
    dy = rad * 0.1564344650;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 2.9845130209103066;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.9914448614;
    dy = rad * 0.1305261922;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 3.0106929596902217;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.9945218954;
    dy = rad * 0.1045284633;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 3.0368728984701368;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.9969173337;
    dy = rad * 0.0784590957;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 3.063052837250052;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.9986295348;
    dy = rad * 0.0523359562;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 3.089232776029967;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.9996573250;
    dy = rad * 0.0261769483;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 3.115412714809882;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -1.0000000000;
    dy = rad * -0.0000000000;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 3.141592653589797;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.9996573250;
    dy = rad * -0.0261769483;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 3.167772592369712;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.9986295348;
    dy = rad * -0.0523359562;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 3.1939525311496273;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.9969173337;
    dy = rad * -0.0784590957;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 3.2201324699295424;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.9945218954;
    dy = rad * -0.1045284633;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 3.2463124087094575;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.9914448614;
    dy = rad * -0.1305261922;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 3.2724923474893726;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.9876883406;
    dy = rad * -0.1564344650;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 3.2986722862692877;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.9832549076;
    dy = rad * -0.1822355255;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 3.3248522250492027;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.9781476007;
    dy = rad * -0.2079116908;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 3.351032163829118;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.9723699204;
    dy = rad * -0.2334453639;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 3.377212102609033;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.9659258263;
    dy = rad * -0.2588190451;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 3.403392041388948;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.9588197349;
    dy = rad * -0.2840153447;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 3.429571980168863;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.9510565163;
    dy = rad * -0.3090169944;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 3.455751918948778;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.9426414911;
    dy = rad * -0.3338068592;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 3.4819318577286933;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.9335804265;
    dy = rad * -0.3583679495;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 3.5081117965086084;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.9238795325;
    dy = rad * -0.3826834324;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 3.5342917352885235;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.9135454576;
    dy = rad * -0.4067366431;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 3.5604716740684386;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.9025852843;
    dy = rad * -0.4305110968;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 3.5866516128483537;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.8910065242;
    dy = rad * -0.4539904997;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 3.6128315516282687;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.8788171127;
    dy = rad * -0.4771587603;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 3.639011490408184;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.8660254038;
    dy = rad * -0.5000000000;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 3.665191429188099;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.8526401644;
    dy = rad * -0.5224985647;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 3.691371367968014;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.8386705679;
    dy = rad * -0.5446390350;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 3.717551306747929;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.8241261886;
    dy = rad * -0.5664062369;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 3.743731245527844;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.8090169944;
    dy = rad * -0.5877852523;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 3.7699111843077593;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.7933533403;
    dy = rad * -0.6087614290;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 3.7960911230876744;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.7771459615;
    dy = rad * -0.6293203910;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 3.8222710618675895;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.7604059656;
    dy = rad * -0.6494480483;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 3.8484510006475046;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.7431448255;
    dy = rad * -0.6691306064;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 3.8746309394274197;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.7253743710;
    dy = rad * -0.6883545757;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 3.9008108782073347;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.7071067812;
    dy = rad * -0.7071067812;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 3.92699081698725;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.6883545757;
    dy = rad * -0.7253743710;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 3.953170755767165;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.6691306064;
    dy = rad * -0.7431448255;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 3.97935069454708;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.6494480483;
    dy = rad * -0.7604059656;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 4.005530633326995;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.6293203910;
    dy = rad * -0.7771459615;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 4.03171057210691;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.6087614290;
    dy = rad * -0.7933533403;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 4.057890510886825;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.5877852523;
    dy = rad * -0.8090169944;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 4.08407044966674;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.5664062369;
    dy = rad * -0.8241261886;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 4.110250388446655;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.5446390350;
    dy = rad * -0.8386705679;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 4.13643032722657;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.5224985647;
    dy = rad * -0.8526401644;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 4.162610266006485;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.5000000000;
    dy = rad * -0.8660254038;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 4.1887902047864;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.4771587603;
    dy = rad * -0.8788171127;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 4.214970143566315;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.4539904997;
    dy = rad * -0.8910065242;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 4.2411500823462305;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.4305110968;
    dy = rad * -0.9025852843;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 4.267330021126146;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.4067366431;
    dy = rad * -0.9135454576;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 4.293509959906061;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.3826834324;
    dy = rad * -0.9238795325;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 4.319689898685976;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.3583679495;
    dy = rad * -0.9335804265;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 4.345869837465891;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.3338068592;
    dy = rad * -0.9426414911;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 4.372049776245806;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.3090169944;
    dy = rad * -0.9510565163;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 4.398229715025721;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.2840153447;
    dy = rad * -0.9588197349;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 4.424409653805636;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.2588190451;
    dy = rad * -0.9659258263;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 4.450589592585551;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.2334453639;
    dy = rad * -0.9723699204;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 4.476769531365466;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.2079116908;
    dy = rad * -0.9781476007;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 4.502949470145381;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.1822355255;
    dy = rad * -0.9832549076;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 4.5291294089252965;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.1564344650;
    dy = rad * -0.9876883406;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 4.555309347705212;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.1305261922;
    dy = rad * -0.9914448614;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 4.581489286485127;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.1045284633;
    dy = rad * -0.9945218954;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 4.607669225265042;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.0784590957;
    dy = rad * -0.9969173337;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 4.633849164044957;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.0523359562;
    dy = rad * -0.9986295348;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 4.660029102824872;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * -0.0261769483;
    dy = rad * -0.9996573250;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 4.686209041604787;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.0000000000;
    dy = rad * -1.0000000000;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 4.712388980384702;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.0261769483;
    dy = rad * -0.9996573250;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 4.738568919164617;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.0523359562;
    dy = rad * -0.9986295348;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 4.764748857944532;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.0784590957;
    dy = rad * -0.9969173337;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 4.790928796724447;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.1045284633;
    dy = rad * -0.9945218954;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 4.8171087355043625;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.1305261922;
    dy = rad * -0.9914448614;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 4.843288674284278;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.1564344650;
    dy = rad * -0.9876883406;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 4.869468613064193;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.1822355255;
    dy = rad * -0.9832549076;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 4.895648551844108;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.2079116908;
    dy = rad * -0.9781476007;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 4.921828490624023;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.2334453639;
    dy = rad * -0.9723699204;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 4.948008429403938;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.2588190451;
    dy = rad * -0.9659258263;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 4.974188368183853;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.2840153447;
    dy = rad * -0.9588197349;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 5.000368306963768;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.3090169944;
    dy = rad * -0.9510565163;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 5.026548245743683;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.3338068592;
    dy = rad * -0.9426414911;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 5.052728184523598;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.3583679495;
    dy = rad * -0.9335804265;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 5.078908123303513;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.3826834324;
    dy = rad * -0.9238795325;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 5.1050880620834285;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.4067366431;
    dy = rad * -0.9135454576;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 5.131268000863344;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.4305110968;
    dy = rad * -0.9025852843;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 5.157447939643259;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.4539904997;
    dy = rad * -0.8910065242;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 5.183627878423174;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.4771587603;
    dy = rad * -0.8788171127;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 5.209807817203089;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.5000000000;
    dy = rad * -0.8660254038;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 5.235987755983004;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.5224985647;
    dy = rad * -0.8526401644;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 5.262167694762919;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.5446390350;
    dy = rad * -0.8386705679;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 5.288347633542834;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.5664062369;
    dy = rad * -0.8241261886;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 5.314527572322749;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.5877852523;
    dy = rad * -0.8090169944;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 5.340707511102664;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.6087614290;
    dy = rad * -0.7933533403;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 5.366887449882579;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.6293203910;
    dy = rad * -0.7771459615;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 5.3930673886624945;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.6494480483;
    dy = rad * -0.7604059656;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 5.41924732744241;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.6691306064;
    dy = rad * -0.7431448255;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 5.445427266222325;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.6883545757;
    dy = rad * -0.7253743710;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 5.47160720500224;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.7071067812;
    dy = rad * -0.7071067812;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 5.497787143782155;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.7253743710;
    dy = rad * -0.6883545757;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 5.52396708256207;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.7431448255;
    dy = rad * -0.6691306064;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 5.550147021341985;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.7604059656;
    dy = rad * -0.6494480483;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 5.5763269601219;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.7771459615;
    dy = rad * -0.6293203910;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 5.602506898901815;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.7933533403;
    dy = rad * -0.6087614290;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 5.62868683768173;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.8090169944;
    dy = rad * -0.5877852523;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 5.654866776461645;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.8241261886;
    dy = rad * -0.5664062369;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 5.6810467152415605;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.8386705679;
    dy = rad * -0.5446390350;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 5.7072266540214756;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.8526401644;
    dy = rad * -0.5224985647;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 5.733406592801391;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.8660254038;
    dy = rad * -0.5000000000;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 5.759586531581306;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.8788171127;
    dy = rad * -0.4771587603;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 5.785766470361221;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.8910065242;
    dy = rad * -0.4539904997;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 5.811946409141136;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.9025852843;
    dy = rad * -0.4305110968;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 5.838126347921051;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.9135454576;
    dy = rad * -0.4067366431;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 5.864306286700966;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.9238795325;
    dy = rad * -0.3826834324;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 5.890486225480881;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.9335804265;
    dy = rad * -0.3583679495;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 5.916666164260796;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.9426414911;
    dy = rad * -0.3338068592;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 5.942846103040711;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.9510565163;
    dy = rad * -0.3090169944;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 5.9690260418206265;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.9588197349;
    dy = rad * -0.2840153447;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 5.9952059806005416;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.9659258263;
    dy = rad * -0.2588190451;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 6.021385919380457;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.9723699204;
    dy = rad * -0.2334453639;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 6.047565858160372;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.9781476007;
    dy = rad * -0.2079116908;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 6.073745796940287;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.9832549076;
    dy = rad * -0.1822355255;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 6.099925735720202;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.9876883406;
    dy = rad * -0.1564344650;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 6.126105674500117;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.9914448614;
    dy = rad * -0.1305261922;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 6.152285613280032;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.9945218954;
    dy = rad * -0.1045284633;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 6.178465552059947;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.9969173337;
    dy = rad * -0.0784590957;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 6.204645490839862;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.9986295348;
    dy = rad * -0.0523359562;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 6.230825429619777;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 0.9996573250;
    dy = rad * -0.0261769483;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 6.2570053683996925;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }



    dx = rad * 1.0000000000;
    dy = rad * 0.0000000000;

    if (tex2D(abs_tex, x + dx, y + dy)  > 20) {
        float grad = PI + tex2D(phi_tex, x + dx, y + dy) - 6.2831853071796075;
        if (-PI / 12.0 < grad && grad < PI / 12.0)
            local_acc++;
    }


    }

    local_acc = blockReduceSum(local_acc * local_acc * local_acc);

    if (threadIdx.x == 0)
        result[width * y + x] = local_acc;

}

#include <stdio.h>

extern "C" void cu_hough(int img_w, int img_h, int pitch, void *gm_abs, void *gm_phi, void *gm_hough)
{
    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipChannelFormatDesc desc = hipCreateChannelDesc<float>();

    hipBindTexture2D(NULL, abs_tex, gm_abs, desc, img_w, img_h, pitch);
    hipBindTexture2D(NULL, phi_tex, gm_phi, desc, img_w, img_h, pitch);

    hipMemset(gm_hough, 0, img_w * img_h * sizeof(int));

    hipEventRecord(start);

    dim3 grid(img_w - MARGIN, img_h - MARGIN);
    dim3 threads(THREAD_COUNT/STEP_SIZE);

    hough_transform<<<grid, threads>>>((int *)gm_hough);
    printf("cuda error: %s\n", hipGetErrorString(hipGetLastError()));

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float elapsed_time = 0;
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf("Took %f ms to compute Hough-Transform\n", elapsed_time);
}
